
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__
void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ 
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for(; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  size_t block_num = 32;
  size_t thread_num = 1024;

  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  checkCuda( hipMallocManaged(&a, size) );
  checkCuda( hipMallocManaged(&b, size) );
  checkCuda( hipMallocManaged(&c, size) );

  initWith<<<block_num, thread_num>>>(3, a, N);
  initWith<<<block_num, thread_num>>>(4, b, N);
  initWith<<<block_num, thread_num>>>(0, c, N);
  
  checkCuda( hipDeviceSynchronize() );

  addVectorsInto<<<block_num, thread_num>>>(c, a, b, N);
  
  checkCuda( hipGetLastError() );
  checkCuda( hipDeviceSynchronize() );

  checkElementsAre(7, c, N);

  checkCuda( hipFree(a) );
  checkCuda( hipFree(b) );
  checkCuda( hipFree(c) );
}
