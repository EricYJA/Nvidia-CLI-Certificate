#include "hip/hip_runtime.h"
#include <stdio.h>

void helloCPU() {
  printf("Hello from the CPU.\n");
}

__global__ 
void helloGPU() {
  printf("Hello from the GPU.\n");
}

__global__ 
void loop() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  printf("%d\n", i);
}

__global__ 
void 

int main() {
  helloGPU<<<1, 1>>>();
  hipDeviceSynchronize();

  helloCPU();

  helloGPU<<<1, 1>>>();   
  hipDeviceSynchronize();

  loop<<<2, 5>>>();
  hipDeviceSynchronize();

  hipError_t syncErr, asyncErr;
  syncErr = hipGetLastError();
  asyncErr = hipDeviceSynchronize();

  if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
  if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));
}
